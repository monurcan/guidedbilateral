#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc.hpp>

#include <iostream>
#include <string.h>
#include <cmath>
#include <chrono>
#include <map>

__global__ void bilateralKernel(int dimx, int dimy, int ncol, unsigned char *orig, unsigned char *guide, int demisize,
								float *sweight, float *iweight, float *gweight,
								float *filtered)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (j >= dimy || i >= dimx)
		return;

	int value, ediff, currentGuide[3], diffGuide;
	float wguide, somme, poids, pixelMoy, currentIntensity, diff, rdiff;
	somme = 1e-6f;
	pixelMoy = 0.0f;
	currentIntensity = filtered[j * dimx + i];
	currentGuide[0] = guide[j * dimx + i];
	if (ncol == 3)
	{
		currentGuide[1] = guide[dimx * dimy + j * dimx + i];
		currentGuide[2] = guide[2 * dimx * dimy + j * dimx + i];
	}
	// don't need to parallize here since only 2x2 max
	for (int k = -demisize; k <= demisize; k++)
	{
		if ((j + k >= 0) && (j + k < dimy))
		{
			for (int l = -demisize; l <= demisize; l++)
			{
				if ((i + l >= 0) && (i + l < dimx))
				{
					value = orig[(j + k) * dimx + i + l];
					diff = fabs((float)value - currentIntensity);
					ediff = (int)floor(diff);
					rdiff = diff - (float)ediff;
					diffGuide = abs(guide[(j + k) * dimx + i + l] - currentGuide[0]);
					wguide = gweight[diffGuide];
					if (ncol == 3)
					{
						diffGuide = abs(guide[dimx * dimy + (j + k) * dimx + i + l] - currentGuide[1]);
						wguide *= gweight[diffGuide];
						diffGuide = abs(guide[2 * dimx * dimy + (j + k) * dimx + i + l] - currentGuide[2]);
						wguide *= gweight[diffGuide];
					}
					poids = ((1.0f - rdiff) * iweight[ediff] + rdiff * iweight[ediff + 1]) * sweight[abs(k)] * sweight[abs(l)] * wguide;
					somme += poids;
					pixelMoy += poids * (float)value;
				}
			}
		}
	}

	filtered[j * dimx + i] = pixelMoy / somme;
}

class GuidedBilateralFilterGPU
{
public:
	// Guided Bilateral Filter parameters
	int hwsize = 2;
	float sscale = 1.5f, iscale = 10.0f, ipower = 0.0f, gscale = 10.0f, gpower = 1.0f;

	// Threshold parameter
	int threshold = 80;

	// Opening parameters
	int morph_size = 1;
	cv::Mat element = getStructuringElement(
		cv::MORPH_ELLIPSE,
		cv::Size(2 * morph_size + 1,
				 2 * morph_size + 1),
		cv::Point(morph_size,
				  morph_size));

	cv::Mat origimg[3], guideimg[3];

	float *filtered_d;
	unsigned char *orig_d;
	unsigned char *guide_d;

	float *sweight, *iweight, *gweight;
	float *sweight_d, *iweight_d, *gweight_d;

	float *filtered_cpu;
	int size_, size;

	GuidedBilateralFilterGPU(int rows, int cols)
	{
		size_ = rows * cols;
		size = size_ * sizeof(float);

		hipMalloc((float **)&filtered_d, size);
		hipMalloc((unsigned char **)&orig_d, size_);
		hipMalloc((unsigned char **)&guide_d, size_);

		sweight = (float *)malloc((hwsize + 1) * sizeof(float));
		hipMalloc((float **)&sweight_d, (hwsize + 1) * sizeof(float));
		hipMalloc((float **)&iweight_d, 257 * sizeof(float));
		hipMalloc((float **)&gweight_d, 256 * sizeof(float));

		filtered_cpu = (float *)malloc(size);
	}

	std::map<std::pair<float, float>, float *> iweights;
	void iweightcalculation(float iscale, float ipower)
	{
		auto ii = iweights.find(std::make_pair(iscale, ipower));
		if (ii != iweights.end())
		{
			iweight = ii->second;
		}
		else
		{
			float *new_iweight = (float *)malloc(257 * sizeof(float));
			/* intensity weight */
			for (int i = 0; i <= 256; i++)
			{
				if (ipower != 1.0f)
					new_iweight[i] = pow(1.0f + (float)(i * i) / (iscale * iscale), ipower - 1.0f);
				else
					new_iweight[i] = 1.0f;
			}
			iweights[std::make_pair(iscale, ipower)] = new_iweight;
			iweight = new_iweight;
		}
	}

	std::map<std::pair<float, float>, float *> gweights;
	void gweightcalculation(float gscale, float gpower)
	{
		auto ii = gweights.find(std::make_pair(gscale, gpower));
		if (ii != gweights.end())
		{
			gweight = ii->second;
		}
		else
		{
			float *new_gweight = (float *)malloc(256 * sizeof(float));
			/* guide weight */
			for (int i = 0; i <= 255; i++)
			{
				if (gpower != 0.0f)
					new_gweight[i] = exp(-(pow(1.0f + (float)(i * i) / (gscale * gscale), gpower) - 1.0f) / gpower);
				else
					new_gweight[i] = 1.0f / (1.0f + (float)(i * i) / (gscale * gscale));
			}
			gweights[std::make_pair(gscale, gpower)] = new_gweight;
			gweight = new_gweight;
		}
	}

	int GuidedBilateralFilterStep(int dimx, int dimy, int ncol, unsigned char *orig, unsigned char *guide, int demisize,
								  float sscale, float iscale, float ipower, float gscale, float gpower)
	{
		for (int i = 0; i <= demisize; i++)
		{
			if (sscale > 0.0f)
				sweight[i] = exp(-0.5f * (float)(i * i) / (sscale * sscale));
			else
				sweight[i] = 1.0f;
		}

		iweightcalculation(iscale, ipower);
		gweightcalculation(gscale, gpower);

		hipMemcpy(sweight_d, sweight, (demisize + 1) * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(iweight_d, iweight, 257 * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(gweight_d, gweight, 256 * sizeof(float), hipMemcpyHostToDevice);

		dim3 block(16, 16);
		dim3 grid((dimx + 15) / 16, (dimy + 15) / 16);
		bilateralKernel<<<grid, block>>>(dimx, dimy, ncol, orig, guide, demisize,
										 sweight_d, iweight_d, gweight_d,
										 filtered_d);

		return (1);
	}

	int GuidedBilateralFilter(int dimx, int dimy, int ncol, unsigned char *orig, unsigned char *guide, int demisize, float sscale, float iscale, float ipower, float gscale, float gpower, unsigned char *result)
	{
		int i, num = 8;

		/* init image */
		for (i = 0; i < dimx * dimy; i++)
			filtered_cpu[i] = (float)(orig[i]);

		hipMemcpy(filtered_d, filtered_cpu, size, hipMemcpyHostToDevice);
		hipMemcpy(orig_d, orig, (dimx * dimy), hipMemcpyHostToDevice);
		hipMemcpy(guide_d, guide, (dimx * dimy), hipMemcpyHostToDevice);

		/* GNC */
		if (ipower <= 1.0f)
		{
			if (!GuidedBilateralFilterStep(dimx, dimy, ncol, orig_d, guide_d, demisize, 0.0, iscale, 1.0, gscale * 5.0, gpower))
				return (0);
			num--;
		}

		if (ipower <= 0.5f)
		{
			if (!GuidedBilateralFilterStep(dimx, dimy, ncol, orig_d, guide_d, demisize, sscale, iscale, 0.5, gscale, gpower))
				return (0);
			num--;
		}

		if (ipower <= 0.0f)
		{
			if (!GuidedBilateralFilterStep(dimx, dimy, ncol, orig_d, guide_d, demisize, sscale, iscale, 0.0, gscale, gpower))
				return (0);
			num--;
		}

		/* final */
		for (i = 0; i < num; i++)
		{
			if (!GuidedBilateralFilterStep(dimx, dimy, ncol, orig_d, guide_d, demisize, sscale, iscale, ipower, gscale, gpower))
				return (0);
		}

		hipMemcpy(filtered_cpu, filtered_d, size, hipMemcpyDeviceToHost);

		for (i = 0; i < dimx * dimy; i++)
			result[i] = (unsigned char)(filtered_cpu[i]);

		// hipError_t error_check = hipGetLastError();printf("%s\n", hipGetErrorString(error_check));

		return (1);
	}

	// very slow implementation, to improve
	// - use cv::cuda functions
	// - do not split and merge the color channels, change the above functions for the images with stacked color channels
	cv::Mat Execute(cv::Mat origimg_, cv::Mat guideimg_)
	{
		// cv::imshow("orig", origimg_);
		// cv::imshow("guide", guideimg_);

		cv::split(origimg_, origimg);
		cv::split(guideimg_, guideimg);

		std::vector<cv::Mat> resultmatIIminusIJ;
		resultmatIIminusIJ.reserve(3);

		// bgr color channels loop
		// TODO: i tried to parallize here, but could not
		for (int i = 0; i < 3; i++)
		{
			unsigned char *resultIJ = new unsigned char[origimg[i].rows * origimg[i].cols];
			unsigned char *resultII = new unsigned char[origimg[i].rows * origimg[i].cols];

			GuidedBilateralFilter(origimg[i].rows, origimg[i].cols, origimg[i].channels(), origimg[i].data, guideimg[i].data, hwsize, sscale, iscale, ipower, gscale, gpower, resultIJ);
			cv::Mat resultmatIJ = cv::Mat(origimg[i].rows, origimg[i].cols, CV_8U, resultIJ);

			GuidedBilateralFilter(origimg[i].rows, origimg[i].cols, origimg[i].channels(), origimg[i].data, origimg[i].data, hwsize, sscale, iscale, ipower, gscale, gpower, resultII);
			cv::Mat resultmatII = cv::Mat(origimg[i].rows, origimg[i].cols, CV_8U, resultII);
			// cv::imwrite("../output_images/result_gpu_IJ.png", resultmatIJ);
			// cv::imwrite("../output_images/result_gpu_II.png", resultmatII);

			cv::Mat resultmatIIminusIJ_channel;
			cv::absdiff(resultmatII, resultmatIJ, resultmatIIminusIJ_channel);

			cv::threshold(resultmatIIminusIJ_channel, resultmatIIminusIJ_channel, threshold, 255, 1);

			morphologyEx(resultmatIIminusIJ_channel, resultmatIIminusIJ_channel,
						 cv::MORPH_OPEN, element,
						 cv::Point(-1, -1), 2);

			resultmatIIminusIJ.emplace_back(resultmatIIminusIJ_channel);

			// cv::imshow("resIJ channel:" + std::to_string(i), resultmatIJ);
			// cv::imshow("resII channel:" + std::to_string(i), resultmatII);
			// cv::imshow("distance channel:" + std::to_string(i), resultmatIIminusIJ_channel);
		}

		cv::Mat mergedresultmatIIminusIJ;
		merge(resultmatIIminusIJ, mergedresultmatIIminusIJ);

		return mergedresultmatIIminusIJ;
	}

	~GuidedBilateralFilterGPU()
	{
		hipFree(filtered_d);
		hipFree(orig_d);
		hipFree(guide_d);

		hipFree(sweight_d);
		hipFree(iweight_d);
		hipFree(gweight_d);

		free(filtered_cpu);

		for(auto ii : iweights) free(ii.second);
		for(auto ii : gweights) free(ii.second);
		free(sweight);
	}
};

int main()
{
	cv::Mat origimg_ = cv::imread("../input_images/makale_1.png", cv::IMREAD_COLOR);
	origimg_.convertTo(origimg_, CV_8U); // just for safety
	cv::Mat guideimg_ = cv::imread("../input_images/makale_0.png", cv::IMREAD_COLOR);
	guideimg_.convertTo(guideimg_, CV_8U); // just for safety

	GuidedBilateralFilterGPU gbFilter(origimg_.rows, origimg_.cols);

	int n_iter = 1;
	auto start = std::chrono::steady_clock::now();
	cv::Mat result;
	for (int i = 0; i < n_iter; i++)
		result = gbFilter.Execute(origimg_, guideimg_);
	auto end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time in milliseconds for one frame: "
			  << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / n_iter
			  << " ms\n";

	// cv::imshow("distance all channels", result);
	// int k = cv::waitKey(0);
	cv::imwrite("../output_images/result_gpu.png", result);

	return 0;
}